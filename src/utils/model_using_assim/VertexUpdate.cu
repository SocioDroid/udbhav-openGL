#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "VertexUpdate.h"

__global__ void arrayUpdateKernel(float *vertexPosition, float *vertexColor, float *initialPosition, bool isDisintegrate, int N, float _speed, int direction, float maxDistance)
{
    if (maxDistance == 0)
        maxDistance = 2.0f;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t state;
    float speed = _speed;
    if (idx < N)
    {
        int i = idx * 3;
        int iColor = idx * 4;
        float px = vertexPosition[i];
        float py = vertexPosition[i + 1];
        float pz = vertexPosition[i + 2];

        hiprand_init(clock64(), i, 0, &state);
        float randomVal = hiprand_uniform(&state);

        // Perform disintegration
        if (isDisintegrate)
        {
            switch (direction)
            {
            case DISINTEGRATE_UP:
            {
                if (py < maxDistance + 1.0f)
                {
                    vertexPosition[i] = px + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                    vertexPosition[i + 1] = py + 8.0 * (randomVal)*speed * 0.1610f;
                    vertexPosition[i + 2] = pz + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                }

                if (py > maxDistance - 2.0f)
                {
                    if (vertexColor[iColor + 3] > 0.0f)
                        vertexColor[iColor + 3] -= speed * 0.5f;
                }
            }
            break;
            case DISINTEGRATE_DOWN:
            {
                if (py > -maxDistance)
                {
                    vertexPosition[i] = px + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                    vertexPosition[i + 1] = py + 8.0 * (0.25 - randomVal) * speed * 0.1610f;
                    vertexPosition[i + 2] = pz + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                }

                if (py < -maxDistance + 3.0f)
                {
                    if (vertexColor[iColor + 3] > 0.0f)
                        vertexColor[iColor + 3] -= speed * 0.5f;
                }
            }
            break;
            case DISINTEGRATE_LEFT:
            {
                if (px > -maxDistance)
                {
                    vertexPosition[i] = px + 8.0 * (0.25 - randomVal) * speed * 0.1610f;
                    vertexPosition[i + 1] = py + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                    vertexPosition[i + 2] = pz + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                }

                if (px < -maxDistance + 1.0f)
                {
                    if (vertexColor[iColor + 3] > 0.0f)
                        vertexColor[iColor + 3] -= speed * 0.5f;
                }
            }
            break;
            case DISINTEGRATE_RIGHT:
            {
                if (px < maxDistance)
                {
                    vertexPosition[i] = px + 8.0 * (randomVal)*speed * 0.1610f;
                    vertexPosition[i + 1] = py + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                    vertexPosition[i + 2] = pz + 1.5 * (0.5 - randomVal) * speed * 0.1610f;
                }

                if (px > maxDistance - 1.0f)
                {
                    if (vertexColor[iColor + 3] > 0.0f)
                        vertexColor[iColor + 3] -= speed * 0.5f;
                }
            }
            break;
            }
        }
        // Perform Integration
        else
        {
            float ix = initialPosition[i];
            float iy = initialPosition[i + 1];
            float iz = initialPosition[i + 2];

            float dx = abs(px - ix);
            float dy = abs(py - iy);
            float dz = abs(pz - iz);

            float d = dx + dy + dx;

            switch (direction)
            {
            case DISINTEGRATE_UP:
            {
                if (d > -1.0)
                {
                    vertexPosition[i] = px - (px - ix) / dx * speed * 0.1610f * (0.85 - randomVal);
                    vertexPosition[i + 1] = py + (iy - py) / dy * speed * 0.1610f * (1.0 + randomVal);
                    vertexPosition[i + 2] = pz - (pz - iz) / dz * speed * 0.1610f * (0.85 - randomVal);
                }

                if (py < maxDistance + 1.0f)
                {
                    if (vertexColor[iColor + 3] < 1.0f)
                        vertexColor[iColor + 3] += speed * 0.5f;
                }
            }
            break;
            case DISINTEGRATE_DOWN:
            {
                if (d > -1.0)
                {
                    vertexPosition[i] = px - (px - ix) / dx * speed * 0.1610f * (0.85 - randomVal);
                    vertexPosition[i + 1] = py - (py - iy) / dy * speed * 0.1610f * (1 + randomVal);
                    vertexPosition[i + 2] = pz - (pz - iz) / dz * speed * 0.1610f * (0.85 - randomVal);
                }

                if (py > -maxDistance)
                {
                    if (vertexColor[iColor + 3] < 1.0f)
                        vertexColor[iColor + 3] += speed * 0.5f;
                }
            }
            break;
            case DISINTEGRATE_LEFT:
            {
                if (d > -1.0)
                {
                    vertexPosition[i] = px - (px - ix) / dx * speed * 0.1610f * (1.0 + randomVal);
                    vertexPosition[i + 1] = py - (py - iy) / dy * speed * 0.1610f * (0.85 - randomVal);
                    vertexPosition[i + 2] = pz - (pz - iz) / dz * speed * 0.1610f * (0.85 - randomVal);
                }

                if (px > -maxDistance)
                {
                    if (vertexColor[iColor + 3] < 1.0f)
                        vertexColor[iColor + 3] += speed * 0.5f;
                }
            }
            break;
            case DISINTEGRATE_RIGHT:
            {
                if (d > -1.0)
                {
                    vertexPosition[i] = px + (ix - px) / dx * speed * 0.1610f * (1.0 + randomVal);
                    vertexPosition[i + 1] = py - (py - iy) / dy * speed * 0.1610f * (0.85 - randomVal);
                    vertexPosition[i + 2] = pz - (pz - iz) / dz * speed * 0.1610f * (0.85 - randomVal);
                }

                if (px < maxDistance - 1.0f)
                {
                    if (vertexColor[iColor + 3] < 1.0f)
                        vertexColor[iColor + 3] += speed * 0.5f;
                }
            }
            break;
            }
        }
    }
}

void launch_arrayUpdateKernel(void *cudaPointerPosition, void *cudaPointerColor, float *initialPosition, bool isDisintegrate, int N, float _speed, int direction, float maxDistance)
{
    dim3 dimGrid = dim3((int)ceil((float)N / 256.0f), 1, 1);
    dim3 dimBlock = dim3(256, 1, 1);
    arrayUpdateKernel<<<dimGrid, dimBlock>>>((float *)cudaPointerPosition, (float *)cudaPointerColor, initialPosition, isDisintegrate, N, _speed, direction, maxDistance);
}

// REDUCE VECTOR SIZE
__global__ void reduceVectorSizeKernel(float *inputArray, int inputSize, float *outputArray, int outputSize, float multiplier)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < outputSize)
    {
        int index = (int)(multiplier * tid);
        if (index < inputSize)
        {
            outputArray[tid * 3] = inputArray[index * 3];
            outputArray[tid * 3 + 1] = inputArray[index * 3 + 1];
            outputArray[tid * 3 + 2] = inputArray[index * 3 + 2];
        }
        else
        {
            outputArray[tid * 3] = inputArray[0];
            outputArray[tid * 3 + 1] = inputArray[1];
            outputArray[tid * 3 + 2] = inputArray[2];
        }
    }
}

void launch_reduceVectorSizeKernel(float *input, float *output, int inputSize, int outputSize)
{
    // Define thread block and grid dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (outputSize + threadsPerBlock - 1) / threadsPerBlock;
    float step = (float)(inputSize - 1) / (outputSize - 1);

    reduceVectorSizeKernel<<<blocksPerGrid, threadsPerBlock>>>(input, inputSize / 3, output, outputSize, step);
}