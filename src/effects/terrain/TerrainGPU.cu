#include "hip/hip_runtime.h"
#include "TerrainGPU.h"

__global__ void generateTileGridKernel(float *positions, float offset_x, float offset_y, int gridLength, float tileW)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < gridLength && j < gridLength)
    {
        float sc = tileW;

        float I_x = 1.0f * sc;
        float I_y = 0.0f;
        float J_x = 0.0f;
        float J_y = 1.0f * sc;

        float pos_x = (float)(j - gridLength / 2) * I_x + (float)(i - gridLength / 2) * J_x;
        float pos_y = (float)(j - gridLength / 2) * I_y + (float)(i - gridLength / 2) * J_y;

        int index = i * gridLength + j;
        positions[index * 2] = pos_x + offset_x;
        positions[index * 2 + 1] = pos_y + offset_y;
    }
}

void generateTileGridGPU(std::vector<std::pair<float, float>> &positionVec, std::pair<float, float> offset, int gridLength, float tileW)
{
    float *d_positions;
    size_t size = positionVec.size() * 2 * sizeof(float);

    hipMalloc(&d_positions, size);

    dim3 blockDim(16, 16);
    dim3 gridDim((gridLength + blockDim.x - 1) / blockDim.x, (gridLength + blockDim.y - 1) / blockDim.y);

    generateTileGridKernel<<<gridDim, blockDim>>>(d_positions, offset.first, offset.second, gridLength, tileW);
    hipDeviceSynchronize();

    hipMemcpy(positionVec.data(), d_positions, size, hipMemcpyDeviceToHost);

    hipFree(d_positions);
}
